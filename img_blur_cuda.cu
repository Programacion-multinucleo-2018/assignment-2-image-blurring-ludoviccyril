#include "hip/hip_runtime.h"
#include "common.h"
#include <chrono>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>

#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

__constant__ int blur_radius = 2;
__constant__ int blur_diameter = 5;
__constant__ double blur_matrix[5][5] = {{0.04, 0.04, 0.04, 0.04, 0.04},
                                  {0.04, 0.04, 0.04, 0.04, 0.04},
                                  {0.04, 0.04, 0.04, 0.04, 0.04},
                                  {0.04, 0.04, 0.04, 0.04, 0.04},
                                  {0.04, 0.04, 0.04, 0.04, 0.04}};

__global__ void blur(unsigned char *original_image, unsigned char *copy_image, int width, int height, int step, int channels) {
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;

  int index = step * i + channels * j;
  copy_image[index] = 0.0;
  copy_image[index + 1] = 0.0;
  copy_image[index + 2] = 0.0;
  for (int k = 0; k < blur_radius * 2 + 1; k++) {
    for (int l = 0; l < blur_radius * 2 + 1; l++) {
      int neighbor_index = step * (i + k - blur_radius) +
                            channels * (j + l - blur_radius);
      copy_image[index] +=
          blur_matrix[k][l] * original_image[neighbor_index];
      copy_image[index + 1] +=
          blur_matrix[k][l] * original_image[neighbor_index + 1];
      copy_image[index + 2] +=
          blur_matrix[k][l] * original_image[neighbor_index + 2];
    }
  }
}

void image_blur(const cv::Mat& input, cv::Mat& output){


	size_t colorBytes = input.step * input.rows;
	size_t blurredBytes = output.step * output.rows;

	unsigned char *d_input, *d_output;

	// Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_input, colorBytes), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<unsigned char>(&d_output, blurredBytes), "CUDA Malloc Failed");

	SAFE_CALL(hipMemcpy(d_input, input.ptr(), colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

  
	const dim3 block(64, 64);

	const dim3 grid((int)ceil((float)input.rows / block.x), (int)ceil((float)input.cols/ block.y));
	printf("image_blur_kernel<<<(%d, %d) , (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);

  auto start_cpu = chrono::high_resolution_clock::now();
	blur <<<grid, block >>>(d_input, d_output, input.cols, input.rows, static_cast<int>(input.step), input.channels());
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");
  auto end_cpu = chrono::high_resolution_clock::now();

	SAFE_CALL(hipMemcpy(output.ptr(), d_output, blurredBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

	SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
  SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
  
  chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;

  cout << "blur <<<(" << grid.x << ", " << grid.y << "), (" << block.x
       << ", " << block.y << ")>>> elapsed " << duration_ms.count()
       << "ms." << endl;
}

int main(int argc, char *argv[]){
  // Get the image path
	string imagePath;
  (argc < 2) ? imagePath = "image.jpg" : imagePath = argv[1];

	Mat input = imread(imagePath, CV_LOAD_IMAGE_COLOR);

	if (input.empty()){
		cout << "Image Not Found!" << std::endl;
		cin.get();
		return -1;
	}

	Mat output = input.clone();

	image_blur(input, output);
  cv::imwrite("output.jpg", output);

	return 0;
}
